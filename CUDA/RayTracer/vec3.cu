#include "hip/hip_runtime.h"
#include "rt.h"

Vec3 operator+(const Vec3 &a, const Vec3 &b)
{
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}

Vec3 operator-(const Vec3 &a, const Vec3 &b)
{
  return {a.x - b.x, a.y - b.y, a.z - b.z};
}

Vec3 operator*(const Vec3 &a, const Vec3 &b)
{
  return {a.x * b.x, a.y * b.y, a.z * b.z};
}

Vec3 operator/(const Vec3 &a, const Vec3 &b)
{
  return {a.x / b.x, a.y / b.y, a.z / b.z};
}

Vec3 cross(const Vec3 &a, const Vec3 &b)
{
  return {a.y * b.z - a.z * b.y, a.z * b.x + a.x * b.z, a.x * b.y - a.y * b.x};
}

float dot(const Vec3 &a, const Vec3 &b)
{
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

float norm_sq(const Vec3 &a)
{
  return dot(a, a);
}

float norm(const Vec3 &a)
{
  return sqrt(norm_sq(a));
}
