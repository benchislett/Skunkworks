#include "hip/hip_runtime.h"
#include "rt.h"

Vec3 operator+(const Vec3 &a, const Vec3 &b)
{
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}

Vec3 operator-(const Vec3 &a, const Vec3 &b)
{
  return {a.x - b.x, a.y - b.y, a.z - b.z};
}

Vec3 operator*(const Vec3 &a, const Vec3 &b)
{
  return {a.x * b.x, a.y * b.y, a.z * b.z};
}

Vec3 operator*(const Vec3 &a, float x)
{
  return {x * a.x, x * a.y, x * a.z};
}

Vec3 operator/(const Vec3 &a, const Vec3 &b)
{
  return {a.x / b.x, a.y / b.y, a.z / b.z};
}

Vec3 operator/(const Vec3 &a, float x)
{
  return {a.x / x, a.y / x, a.z / x};
}

bool operator==(const Vec3 &a, const Vec3 &b)
{
  return EQ(a.x, b.x) && EQ(a.y, b.y) && EQ(a.z, b.z);
}

std::ostream& operator<<(std::ostream& os, const Vec3 &a)
{
  os << "Vec3(" << a.x << ", " << a.y << ", " << a.z << ")";
  return os;
}

Vec3 cross(const Vec3 &a, const Vec3 &b)
{
  return {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x};
}

float dot(const Vec3 &a, const Vec3 &b)
{
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

float norm_sq(const Vec3 &a)
{
  return dot(a, a);
}

float norm(const Vec3 &a)
{
  return sqrt(norm_sq(a));
}

Vec3 unit(const Vec3 &a)
{
  return a / norm(a);
}

void make_unit(Vec3 *a)
{
  float n = norm(*a);
  a->x /= n;
  a->y /= n;
  a->z /= n;
}
