#include "hip/hip_runtime.h"
#include "rt.cuh"

__device__ Vec3 get_color(const Ray &r, const World &w, const RenderParams &p)
{
  HitData rec;
  Vec3 color = p.background;
  Ray ray = r;

  while (hit(ray, w, &rec)) {
    ray = {rec.point, rec.normal};
    color = color * 0.5;
  }
  return color;
}

__global__ void render_kernel(float *out, const World w, const RenderParams p)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
 
  if ((i >= p.width) || (j >= p.height)) return;

  int idx = 3 * (i + p.width * j);
  float u = (float)i / (float)p.width;
  float v = (float)j / (float)p.height;

  Ray r = get_ray(p.cam, u, v);
  Vec3 color = get_color(r, w, p);

  out[idx + 0] = color.x;
  out[idx + 1] = color.y;
  out[idx + 2] = color.z;
}

void render(float *host_out, const RenderParams &p, World w)
{
  float *device_out;
  hipMalloc((void **)&device_out, 3 * p.width * p.height * sizeof(float));

  Tri *device_tris;
  hipMalloc((void **)&device_tris, w.n * sizeof(Tri));
  hipMemcpy(device_tris, w.t, w.n * sizeof(Tri), hipMemcpyHostToDevice);
  w.t = device_tris;

  int tx = 8;
  int ty = 8;

  dim3 blocks(p.width / tx + 1, p.height / ty + 1);
  dim3 threads(tx, ty);

  render_kernel<<<blocks, threads>>>(device_out, w, p);

  hipDeviceSynchronize();

  hipMemcpy(host_out, device_out, 3 * p.width * p.height * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_out);
  hipFree(device_tris);
}
