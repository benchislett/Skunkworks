#include "hip/hip_runtime.h"
#include "rt.cuh"

__device__ Vec3 get_color(const Ray &r, const Vec3 &background)
{
  return background;
}

__global__ void render_kernel(float *out, const RenderParams p)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
 
  if ((i >= p.width) || (j >= p.height)) return;

  int idx = 3 * (i + p.width * j);
  float u = (float)i / (float)p.width;
  float v = (float)j / (float)p.height;

  Ray r = get_ray(p.cam, u, v);
  Vec3 color = get_color(r, p.background);

  out[idx + 0] = color.x;
  out[idx + 1] = color.y;
  out[idx + 2] = color.z;
}

void render(float *host_out, const RenderParams &p)
{
  float *device_out;
  hipMalloc((void **)&device_out, 3 * p.width * p.height * sizeof(float));

  int tx = 8;
  int ty = 8;

  dim3 blocks(p.width / tx + 1, p.height / ty + 1);
  dim3 threads(tx, ty);

  render_kernel<<<blocks, threads>>>(device_out, p);

  hipDeviceSynchronize();

  hipMemcpy(host_out, device_out, 3 * p.width * p.height * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_out);
}
