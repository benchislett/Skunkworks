#include "hip/hip_runtime.h"
#include "rt.cuh"
#define MAX_DEPTH 16

inline __device__ Vec3 lerp(const Vec3 &a, const Vec3 &b, float factor) {
  return a * (1 - factor) + b * factor;
}

__device__ Vec3 get_color(const Ray &r, const BVHWorld &w, const RenderParams &p, hiprandState *rand_state)
{
  Vec3 color = {1.0, 1.0, 1.0};
  const Vec3 white = {1.0, 1.0, 1.0};
  HitData rec;
  Ray ray = r;
  int depth = 0;

  Vec3 attenuation = {0.75, 0.75, 0.75};

  while (hit(ray, w, &rec)) {
    ray.from = rec.point;
    ray.d = rec.normal + random_in_unit_sphere(rand_state);
    color = color * attenuation;
    if (depth++ >= MAX_DEPTH) {
      color = {0.0, 0.0, 0.0};
      break;
    }
  }

  float t = 0.5 * (unit(ray.d).y + 1);
  color = color * lerp(white, p.background, t);

  return color;
}

__global__ void render_kernel(float *out, const BVHWorld w, const RenderParams p, hiprandState *rand_state)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
 
  if ((i >= p.width) || (j >= p.height)) return;

  int idx = 3 * (i + p.width * j);
  hiprandState local_rand_state = rand_state[idx / 3];

  Vec3 color = {0.0, 0.0, 0.0};
  for (int c = 0; c < p.samples; c++)
  {
    float irand = (float)i + hiprand_uniform(&local_rand_state);
    float jrand = (float)j + hiprand_uniform(&local_rand_state);

    float u = irand / (float)p.width;
    float v = jrand / (float)p.height;

    Ray r = get_ray(p.cam, u, v);
    color = color + get_color(r, w, p, &local_rand_state);
  }
  color = color / (float)p.samples;

  out[idx + 0] = color.x;
  out[idx + 1] = color.y;
  out[idx + 2] = color.z;
}

void render(float *host_out, const RenderParams &p, World w)
{
  int imgsize = 3 * p.width * p.height;
  int tx = 16;
  int ty = 16;

  float *device_out;
  hipMalloc((void **)&device_out, imgsize * sizeof(float));

  Tri *device_tris;
  hipMallocManaged((void **)&device_tris, w.n * sizeof(Tri));
  hipMemcpy(device_tris, w.t, w.n * sizeof(Tri), hipMemcpyHostToDevice);

  uint64_t *morton_codes;
  hipMallocManaged((void **)&morton_codes, w.n * sizeof(uint64_t));
  populate_morton_codes<<<w.n / tx + 1, tx>>>(device_tris, morton_codes, w.n, w.bounds, 2097151);

  thrust::sort_by_key(thrust::device, morton_codes, morton_codes + w.n, device_tris);

  BoundingNode *device_nodes;
  hipMallocManaged((void **)&device_nodes, 2 * w.n * sizeof(BoundingNode));

  hipDeviceSynchronize();

  int acc = 2 * w.n;
  while (acc > 0) {
    populate_bvh<<<acc / tx + 1, tx>>>(device_tris, device_nodes, w.n, 2 * w.n, acc / 2, acc);

    acc /= 2;
    hipDeviceSynchronize();
  }

  BVHWorld bw = {w.n, 2 * w.n, device_nodes};

  dim3 blocks(p.width / tx + 1, p.height / ty + 1);
  dim3 threads(tx, ty);

  hiprandState *rand_state;
  hipMalloc((void **)&rand_state, imgsize * sizeof(hiprandState));

  rand_init<<<blocks, threads>>>(p, rand_state);
  render_kernel<<<blocks, threads>>>(device_out, bw, p, rand_state);

  hipDeviceSynchronize();

  hipMemcpy(host_out, device_out, imgsize * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_out);
  hipFree(device_tris);
  hipFree(morton_codes);
}
