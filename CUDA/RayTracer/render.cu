#include "hip/hip_runtime.h"
#include "rt.cuh"
#define MAX_DEPTH 1

__device__ Vec3 get_color(const Ray &r, const BVHWorld &w, const RenderParams &p, hiprandState *rand_state)
{
  Vec3 color = {1.0, 1.0, 1.0};
  const Vec3 white = {1.0, 1.0, 1.0};
  Vec3 tri_color = {0.9, 0.5, 0.7};
  HitData rec = {-1.0, white, white};
  Ray ray = r;
  int depth = 1;

  while (hit(ray, w, &rec)) {
    ray.from = rec.point;
    ray.d = rec.normal;
    color = color * tri_color;
    if (depth++ >= MAX_DEPTH) break;
  }

  float t = 0.5 * (unit(ray.d).y + 1);
  color = color * ((white * (1.0-t)) + (p.background * t));

  return color;
}

__global__ void render_kernel(float *out, const BVHWorld w, const RenderParams p, hiprandState *rand_state)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
 
  if ((i >= p.width) || (j >= p.height)) return;

  int idx = 3 * (i + p.width * j);
  hiprandState local_rand_state = rand_state[idx / 3];

  Vec3 color = {0.0, 0.0, 0.0};
  for (int c = 0; c < p.samples; c++)
  {
    float irand = (float)i + hiprand_uniform(&local_rand_state);
    float jrand = (float)j + hiprand_uniform(&local_rand_state);

    float u = irand / (float)p.width;
    float v = jrand / (float)p.height;

    Ray r = get_ray(p.cam, u, v);
    color = color + get_color(r, w, p, &local_rand_state);
  }
  color = color / (float)p.samples;

  out[idx + 0] = color.x;
  out[idx + 1] = color.y;
  out[idx + 2] = color.z;
}

__global__ void populate_bvh(Tri *t, BoundingNode *nodes, int n, int bn, int lower, int upper) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i < lower || i >= upper) return;

  int left = 2 * i + 1;
  int right = 2 * i + 2;

  if (left < bn && right < bn) {
    nodes[i].left = &nodes[left];
    nodes[i].right = &nodes[right];
    nodes[i].slab = bounding_slab(nodes[left].slab, nodes[right].slab);
    nodes[i].t = NULL;
  } else if (left < bn) {
    nodes[i].left = &nodes[left];
    nodes[i].right = NULL;
    nodes[i].slab = nodes[left].slab;
    nodes[i].t = NULL;
  } else if (right < bn) {
    nodes[i].left = NULL;
    nodes[i].right = &nodes[right];
    nodes[i].slab = nodes[right].slab;
    nodes[i].t = NULL;
  } else {
    nodes[i].left = NULL;
    nodes[i].right = NULL;
    nodes[i].slab = bounding_slab(t[i - n]);
    nodes[i].t = &t[i - n];
  }
}

void render(float *host_out, const RenderParams &p, World w)
{
  int imgsize = 3 * p.width * p.height;

  float *device_out;
  hipMalloc((void **)&device_out, imgsize * sizeof(float));

  Tri *device_tris;
  hipMallocManaged((void **)&device_tris, w.n * sizeof(Tri));
  hipMemcpy(device_tris, w.t, w.n * sizeof(Tri), hipMemcpyHostToDevice);
  w.t = device_tris;

  BoundingNode *device_nodes;
  hipMallocManaged((void **)&device_nodes, 2 * w.n * sizeof(BoundingNode));

  int tx = 8;
  int ty = 8;

  hipDeviceSynchronize();

  int acc = 2 * w.n;
  while (acc > 0) {
    populate_bvh<<<acc / tx + 1, tx>>>(device_tris, device_nodes, w.n, 2 * w.n, acc / 2, acc);

    acc /= 2;
    hipDeviceSynchronize();
  }

  BVHWorld bw = {w.n, 2 * w.n, device_nodes};

  dim3 blocks(p.width / tx + 1, p.height / ty + 1);
  dim3 threads(tx, ty);

  hiprandState *rand_state;
  hipMalloc((void **)&rand_state, imgsize * sizeof(hiprandState));

  rand_init<<<blocks, threads>>>(p, rand_state);
  render_kernel<<<blocks, threads>>>(device_out, bw, p, rand_state);

  hipDeviceSynchronize();

  hipMemcpy(host_out, device_out, imgsize * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_out);
  hipFree(device_tris);
}
